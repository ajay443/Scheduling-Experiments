

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

/* Program Parameters */
#define MAXN 15000  /* Max value of N */
#define TILE_WIDTH 32  /* Width of each block */
int N;  /* Matrix size */

/* Matrices */
float overall;
char buffer[10000];
  char *pbuff;
  int *classIdArray = (int *)malloc(sizeof(int)*26);
  int *groupIdArray = (int *)malloc(sizeof(int)*26);
  int *roomIdArray =(int *) malloc(sizeof(int)*26);
  int *roomSizeAsArray = (int *)malloc(sizeof(int)*5);
  int *groupSizeAsArray = (int *)malloc(sizeof(int)*11);
  int *timeSlotIdArray = (int *)malloc(sizeof(int)*26);
  int *profIdArray = (int *)malloc(sizeof(int)*26);
  int *clashes=(int *)calloc(26*26,sizeof(int));

int   *dclassIdArray,*dgroupIdArray,*droomIdArray,*droomSizeAsArray,*dgroupSizeAsArray,*dtimeSlotIdArray,*dprofIdArray,*dclashes;

/* junk */
#define randm() 4|2[uid]&3


/* returns a seed for srand based on the time */
unsigned int time_seed() {
  struct timeval t;
  struct timezone tzdummy;

  gettimeofday(&t, &tzdummy);
  return (unsigned int)(t.tv_usec);
}

//Kernel to calculate fitness

__global__ void calculateFitness(int *classIds, int *roomIds, int *roomCapacities, int *groupIds, int *groupSizes, int *timeSlotIds, int *profIds, int *clashes)
{
	int i = threadIdx.x;
	int j = blockIdx.x ;
	
	if(classIds[i]==classIds[j] && roomCapacities[classIds[i]]<groupSizes[groupIds[classIds[i]]])
		clashes[i*j+i]++;
		
	if (roomIds[classIds[j]] == roomIds[classIds[i]] && timeSlotIds[classIds[i]] == timeSlotIds[classIds[j]]
			&& classIds[i] != classIds[j]) {
		clashes[i*j+i]++;
	}

	if (profIds[classIds[j]] == profIds[classIds[j]] && timeSlotIds[classIds[i]] == timeSlotIds[classIds[j]]
			&& classIds[i] != classIds[j]) {
		clashes[i*j+i]++;
	}
}

//Main function to execute fitness function 

int main(int argc, char **argv) {
  /* Timing variables */
  struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
  struct timezone tzdummy;
  clock_t etstart2, etstop2;  /* Elapsed times using times() */
  unsigned long long usecstart, usecstop;
  struct tms cputstart, cputstop;  /* CPU times for my processes */
  /* Process program parameters */
  srand(time_seed());
  int size = N * N * sizeof( int );

  
  /* --------Reading frm intermediate data ----------*/

  
   FILE *fp;
  int i=0;
  fp=fopen("file.txt","r");
  
  while (1) {
    if (!fgets(buffer, sizeof buffer, fp)||feof(fp)) break;
    pbuff = buffer;
    i=0;
    /* copying from intermediate file*/
    while (1) {
      if (*pbuff == 13 || *pbuff == 10) break;
      classIdArray[i] = strtol(pbuff, &pbuff, 10);
      //printf(" %d", classIdArray[i]);
      i++;
    }
    //printf("\n");
    i=0;
    //printf("groupIdArray\n");
    fgets(buffer, sizeof buffer, fp);
    pbuff = buffer;
    while (1) {
      if (*pbuff == 13 || *pbuff == 10) break;
      groupIdArray[i] = strtol(pbuff, &pbuff, 10);
      //printf(" %d", groupIdArray[i]);
      i++;
    }

    fgets(buffer, sizeof buffer, fp);
    pbuff = buffer;
    i=0;

    while (1) {
      if (*pbuff == 13 || *pbuff == 10) break;
      roomIdArray[i] = strtol(pbuff, &pbuff, 10);
      //printf(" %d", roomIdArray[i]);
      i++;
    }

    fgets(buffer, sizeof buffer, fp);
    pbuff = buffer;
    i=0;

    while (1) {
      if (*pbuff == 13 || *pbuff == 10) break;
      roomSizeAsArray[i] = strtol(pbuff, &pbuff, 10);
      //printf(" %d", roomSizeAsArray[i]);
      i++;
    }

    fgets(buffer, sizeof buffer, fp);
    pbuff = buffer;
    i=0;
    while (1) {
      if (*pbuff == 13 || *pbuff == 10) break;
      groupSizeAsArray[i] = strtol(pbuff, &pbuff, 10);
       i++;
    }
    i=0;
     fgets(buffer, sizeof buffer, fp);
    pbuff = buffer;
     while (1) {
      if (*pbuff == 13 || *pbuff == 10) break;
      timeSlotIdArray[i] = strtol(pbuff, &pbuff, 10);
       i++;
    }
     fgets(buffer, sizeof buffer, fp);
    pbuff = buffer;
    i=0;
     while (1) {
      if (*pbuff == 13 || *pbuff == 10) break;
      profIdArray[i] = strtol(pbuff, &pbuff, 10);
       i++;
    }
   
  

  
  /* */

 
  hipMalloc( (void **) &dclashes, sizeof(int)*26*26 );
  hipMalloc( (void **) &dclassIdArray, sizeof(int)*26 );
  hipMalloc( (void **) &dgroupIdArray, sizeof(int)*26 );
  hipMalloc( (void **) &droomIdArray, sizeof(int)*26 );
  hipMalloc( (void **) &droomSizeAsArray, sizeof(int)*5 );
  hipMalloc( (void **) &dgroupSizeAsArray, sizeof(int)*11 );
  hipMalloc( (void **) &dtimeSlotIdArray, sizeof(int)*26 );
  hipMalloc( (void **) &dprofIdArray, sizeof(int)*26 );
  dim3 dimGrid(26,1,1);
  dim3 dimBlock(26, 1,1);
  hipMemcpy( dclassIdArray, classIdArray, sizeof(int)*26, hipMemcpyHostToDevice );
  hipMemcpy( dgroupIdArray, groupIdArray, sizeof(int)*26, hipMemcpyHostToDevice );
  hipMemcpy( droomIdArray, roomIdArray, sizeof(int)*26, hipMemcpyHostToDevice );
  hipMemcpy( droomSizeAsArray, roomSizeAsArray, sizeof(int)*5, hipMemcpyHostToDevice );
  hipMemcpy( dgroupSizeAsArray, groupSizeAsArray, sizeof(int)*11, hipMemcpyHostToDevice );
  hipMemcpy( dtimeSlotIdArray, timeSlotIdArray, sizeof(int)*26, hipMemcpyHostToDevice );
  hipMemcpy( dprofIdArray, profIdArray, sizeof(int)*26, hipMemcpyHostToDevice );
	
	  /* Start Clock */
  gettimeofday(&etstart, &tzdummy);
  etstart2 = times(&cputstart);

  calculateFitness<<<dimGrid,dimBlock>>>(dclassIdArray,droomIdArray,droomSizeAsArray,dgroupIdArray,dgroupSizeAsArray,dtimeSlotIdArray,dprofIdArray,dclashes);

	
  /* Stop Clock */
  gettimeofday(&etstop, &tzdummy);
  etstop2 = times(&cputstop);


  hipMemcpy( clashes, dclashes, sizeof(int)*26*26, hipMemcpyDeviceToHost );



  usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
  usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

  int x=0;
  int y=26*26;
  int z=0;
  for(x=0;x<y;x++)
  {
	  z=z+clashes[x];
  }

  
  hipFree(dclashes);
  hipFree(dclassIdArray);
  hipFree(dgroupIdArray);
  hipFree(droomIdArray);
  hipFree(droomSizeAsArray);
  hipFree(dgroupSizeAsArray);
  hipFree(dtimeSlotIdArray);
  hipFree(dprofIdArray);
  overall=overall+(float)(usecstop - usecstart)/(float)1000;
  
  
}
free(classIdArray);
  free(groupIdArray);
  free(roomIdArray);
  free(roomSizeAsArray);
  free(clashes);
  free(groupSizeAsArray);
  free(timeSlotIdArray);
  free(profIdArray); 

  printf("\nElapsed time = %g ms.\n",overall);
  exit(0);
}


